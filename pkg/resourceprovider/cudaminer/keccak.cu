#include "hip/hip_runtime.h"
/*
 * keccak.cu  Implementation of Keccak/SHA3 digest
 *
 * Date: 12 June 2019
 * Revision: 1
 *
 * This file is released into the Public Domain.
 */

extern "C"
{
#include "keccak.cuh"
}

#define KECCAK_ROUND 24
#define KECCAK_STATE_SIZE 25
#define KECCAK_Q_SIZE 192

typedef union
{
    uint2 uint2;
    uint64_t uint64;
    uint8_t uint8[8];
} nonce_t;

__constant__ uint64_t d_midstate[25];
__constant__ uint64_t d_target[1];

__device__ __forceinline__ nonce_t bswap_64(nonce_t const input)
{
    nonce_t output;
    asm("{"
        "  prmt.b32 %0, %3, 0, 0x0123;"
        "  prmt.b32 %1, %2, 0, 0x0123;"
        "}" : "=r"(output.uint2.x), "=r"(output.uint2.y) : "r"(input.uint2.x), "r"(input.uint2.y));
    return output;
}

__device__ __forceinline__ nonce_t xor5(nonce_t const a, nonce_t const b, nonce_t const c, nonce_t const d, nonce_t const e)
{
    nonce_t output;
    asm("{"
        "  lop3.b32 %0, %2, %4, %6, 0x96;"
        "  lop3.b32 %1, %3, %5, %7, 0x96;"
        "  lop3.b32 %0, %0, %8, %10, 0x96;"
        "  lop3.b32 %1, %1, %9, %11, 0x96;"
        "}" : "=r"(output.uint2.x), "=r"(output.uint2.y)
        : "r"(a.uint2.x), "r"(a.uint2.y), "r"(b.uint2.x), "r"(b.uint2.y), "r"(c.uint2.x), "r"(c.uint2.y), "r"(d.uint2.x), "r"(d.uint2.y), "r"(e.uint2.x), "r"(e.uint2.y));
    return output;
}

__device__ __forceinline__ nonce_t xor3(nonce_t const a, nonce_t const b, nonce_t const c)
{
    nonce_t output;
    asm("{"
        "  lop3.b32 %0, %2, %4, %6, 0x96;"
        "  lop3.b32 %1, %3, %5, %7, 0x96;"
        "}" : "=r"(output.uint2.x), "=r"(output.uint2.y)
        : "r"(a.uint2.x), "r"(a.uint2.y), "r"(b.uint2.x), "r"(b.uint2.y), "r"(c.uint2.x), "r"(c.uint2.y));
    return output;
}

__device__ __forceinline__ nonce_t chi(nonce_t const a, nonce_t const b, nonce_t const c)
{
    nonce_t output;
    asm("{"
        "  lop3.b32 %0, %2, %4, %6, 0xD2;"
        "  lop3.b32 %1, %3, %5, %7, 0xD2;"
        "}" : "=r"(output.uint2.x), "=r"(output.uint2.y)
        : "r"(a.uint2.x), "r"(a.uint2.y), "r"(b.uint2.x), "r"(b.uint2.y), "r"(c.uint2.x), "r"(c.uint2.y));

    return output;
}

__device__ __forceinline__ nonce_t rotl(nonce_t input, uint32_t const offset)
{
    asm("{"
        "  .reg .b32 tmp;"
        "  shf.l.wrap.b32 tmp, %1, %0, %2;"
        "  shf.l.wrap.b32 %1, %0, %1, %2;"
        "  mov.b32 %0, tmp;"
        "}" : "+r"(input.uint2.x), "+r"(input.uint2.y) : "r"(offset));
    return input;
}

__device__ __forceinline__ nonce_t rotr(nonce_t input, uint32_t const offset)
{
    asm("{"
        "  .reg .b32 tmp;"
        "  shf.r.wrap.b32 tmp, %0, %1, %2;"
        "  shf.r.wrap.b32 %1, %1, %0, %2;"
        "  mov.b32 %0, tmp;"
        "}" : "+r"(input.uint2.x), "+r"(input.uint2.y) : "r"(offset));
    return input;
}

__device__ uint64_t rotate(uint64_t val, unsigned n) { return val << n | val >> (64 - n); }

// Array of indices and rotation values for P and Pi phases.
__constant__ uint8_t g_ppi_aux[25][2] = {
    {0, 0}, {6, 44}, {12, 43}, {18, 21}, {24, 14}, {3, 28}, {9, 20}, {10, 3}, {16, 45}, {22, 61}, {1, 1}, {7, 6}, {13, 25}, {19, 8}, {20, 18}, {4, 27}, {5, 36}, {11, 10}, {17, 15}, {23, 56}, {2, 62}, {8, 55}, {14, 39}, {15, 41}, {21, 2}};

// Array of indices for ksi phase.
__constant__ uint8_t g_ksi_aux[25][2] = {
    {1, 2}, {2, 3}, {3, 4}, {4, 0}, {0, 1}, {6, 7}, {7, 8}, {8, 9}, {9, 5}, {5, 6}, {11, 12}, {12, 13}, {13, 14}, {14, 10}, {10, 11}, {16, 17}, {17, 18}, {18, 19}, {19, 15}, {15, 16}, {21, 22}, {22, 23}, {23, 24}, {24, 20}, {20, 21}};

__constant__ uint64_t g_iota_aux[24] = {
    0x0000000000000001L, 0x0000000000008082L, 0x800000000000808aL, 0x8000000080008000L, 0x000000000000808bL,
    0x0000000080000001L, 0x8000000080008081L, 0x8000000000008009L, 0x000000000000008aL, 0x0000000000000088L,
    0x0000000080008009L, 0x000000008000000aL, 0x000000008000808bL, 0x800000000000008bL, 0x8000000000008089L,
    0x8000000000008003L, 0x8000000000008002L, 0x8000000000000080L, 0x000000000000800aL, 0x800000008000000aL,
    0x8000000080008081L, 0x8000000000008080L, 0x0000000080000001L, 0x8000000080008008L};

__device__ static void cuda_keccak_permutations(nonce_t *A, nonce_t *C, const int threadIndexInWrap)
{
    size_t s = threadIndexInWrap % 5;
#pragma unroll
    for (int round_idx = 0; round_idx < 24; ++round_idx)
    {
        // Thetta phase.
        C[threadIndexInWrap] = xor5(A[s], A[s + 5], A[s + 10], A[s + 15], A[s + 20]);
        A[threadIndexInWrap] = xor3(A[threadIndexInWrap], C[s + 5 - 1], rotl(C[s + 1], 1));

        // P and Pi combined phases.
        C[threadIndexInWrap].uint64 = rotate(A[g_ppi_aux[threadIndexInWrap][0]].uint64, g_ppi_aux[threadIndexInWrap][1]);

        // Ksi phase.
        A[threadIndexInWrap] = chi(C[threadIndexInWrap], C[g_ksi_aux[threadIndexInWrap][0]], C[g_ksi_aux[threadIndexInWrap][1]]);

        // Iota phase.
        if (threadIndexInWrap == 0)
        {
            A[threadIndexInWrap].uint64 ^= g_iota_aux[round_idx];
        }
    }
}

__device__ static bool hashbelowtarget(const uint8_t *const __restrict__ hash, const uint8_t *const __restrict__ target)
{
    for (int i = 0; i < 32; i++)
    {
        if (hash[i] < target[i])
        {
            return true;
        }
        else if (hash[i] > target[i])
        {
            return false;
        }
    }
    return false;
}

__device__ __noinline__ void addUint256(nonce_t *result, const uint64_t *a, uint64_t b)
{
    uint64_t sum = a[0] + b;
    result[0].uint64 = sum;

    uint64_t carry = (sum < a[0]) ? 1 : 0;
    for (int i = 1; i < 4; i++)
    {
        sum = a[i] + carry;
        result[i].uint64 = sum;
        carry = (sum < a[i]) ? 1 : 0;
    }
}


#define WRAP_IN_BLOCK 32 // equal to block_size/32

extern "C" __global__ __launch_bounds__(1024) void kernel_lilypad_pow(
    const uint8_t *__restrict__ challenge,
    const uint64_t *__restrict__ startNonce,
    const uint8_t *__restrict__ target,
    const uint32_t n_batch,
    const uint32_t hashPerThread, uint8_t *resNonce)
{
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread >= n_batch) // batch must equal with grid*block
    {
        return;
    }

    uint64_t wrapInOneLaunch = thread / 32;
    int threadIndexInWrap = thread % 32; // index in wrap
    if (threadIndexInWrap >= 25)         // abort 26-32 thread
    {
        return;
    }

    int wrapIndexInBlock = threadIdx.x / 32; // one wrap one worker, 25/32 usages

    __shared__ nonce_t stateInBlock[WRAP_IN_BLOCK][KECCAK_STATE_SIZE];
    __shared__ nonce_t cInBlock[WRAP_IN_BLOCK][25];

    nonce_t *state = stateInBlock[wrapIndexInBlock];
    nonce_t *C = cInBlock[wrapIndexInBlock];

    C[threadIndexInWrap].uint64 = 0;

    __syncwarp();
    uint64_t nonceOffset = wrapInOneLaunch * hashPerThread;
    uint64_t endNonceOffset = (wrapInOneLaunch + 1) * hashPerThread;
    for (; nonceOffset < endNonceOffset; nonceOffset++)
    {
        nonce_t nonce[4];

        state[threadIndexInWrap].uint64 = 0;
        if (threadIndexInWrap == 0)
        {
            // increase nonce
            addUint256(nonce, startNonce, nonceOffset);
            memcpy(state, challenge, 32); // Copy challenge into state
            memcpy(state + 4, nonce, 32); // Copy nonce into state starting from index 4

            state[8].uint64 ^= 1;
            state[16].uint64 ^= 9223372036854775808ULL;
        }

        __syncwarp();
        cuda_keccak_permutations(state, C, threadIndexInWrap);

        if (threadIndexInWrap == 0)
        {

            if (hashbelowtarget(state->uint8, target))
            {
                memcpy(resNonce, nonce, 32);
            }

            delete nonce; // 45
        }
    }
}

extern "C" __global__ __launch_bounds__(1024) void kernel_lilypad_pow_debug(
    const uint8_t *__restrict__ challenge,
    const uint64_t *__restrict__ startNonce,
    const uint8_t *__restrict__ target,
    const uint32_t n_batch,
    const uint32_t hashPerThread, uint8_t *resNonce, uint8_t *hash, uint8_t *pack)
{
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread >= n_batch) // batch must equal with grid*block
    {
        return;
    }

    uint64_t wrapInOneLaunch = thread / 32;
    int threadIndexInWrap = thread % 32; // index in wrap
    if (threadIndexInWrap >= 25)         // abort 26-32 thread
    {
        return;
    }

    int wrapIndexInBlock = threadIdx.x / 32; // one wrap one worker, 25/32 usages

    __shared__ nonce_t stateInBlock[WRAP_IN_BLOCK][KECCAK_STATE_SIZE];
    __shared__ nonce_t cInBlock[WRAP_IN_BLOCK][25];

    nonce_t *state = stateInBlock[wrapIndexInBlock];
    nonce_t *C = cInBlock[wrapIndexInBlock];

    C[threadIndexInWrap].uint64 = 0;

    __syncwarp();
    uint64_t nonceOffset = wrapInOneLaunch * hashPerThread;
    uint64_t endNonceOffset = (wrapInOneLaunch + 1) * hashPerThread;
    for (; nonceOffset < endNonceOffset; nonceOffset++)
    {
        uint8_t cuda_pack[64];
        nonce_t nonce[4];

        state[threadIndexInWrap].uint64 = 0;
        if (threadIndexInWrap == 0)
        {
            // increase nonce
            addUint256(nonce, startNonce, nonceOffset);
            memcpy(state, challenge, 32); // Copy challenge into state
            memcpy(state + 4, nonce, 32); // Copy nonce into state starting from index 4

            memcpy(cuda_pack, state, 64);

            state[8].uint64 ^= 1;
            state[16].uint64 ^= 9223372036854775808ULL;
        }

        __syncwarp();
        cuda_keccak_permutations(state, C, threadIndexInWrap);

        if (threadIndexInWrap == 0)
        {

            if (hashbelowtarget(state->uint8, target))
            {
                memcpy(hash, state, 32);
                memcpy(pack, cuda_pack, 64);
                memcpy(resNonce, nonce, 32);
            }

            delete nonce; // 45
        }
    }
}
