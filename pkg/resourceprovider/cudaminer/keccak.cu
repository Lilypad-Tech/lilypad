#include "hip/hip_runtime.h"
/*
 * keccak.cu  Implementation of Keccak/SHA3 digest
 *
 * Date: 12 June 2019
 * Revision: 1
 *
 * This file is released into the Public Domain.
 */
 
 
extern "C"
{
    #include "keccak.cuh"
}

#define KECCAK_ROUND 24
#define KECCAK_STATE_SIZE 25
#define KECCAK_Q_SIZE 192

__constant__ uint64_t CUDA_KECCAK_CONSTS[24] = { 0x0000000000000001, 0x0000000000008082,
                                          0x800000000000808a, 0x8000000080008000, 0x000000000000808b, 0x0000000080000001, 0x8000000080008081,
                                          0x8000000000008009, 0x000000000000008a, 0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
                                          0x000000008000808b, 0x800000000000008b, 0x8000000000008089, 0x8000000000008003, 0x8000000000008002,
                                          0x8000000000000080, 0x000000000000800a, 0x800000008000000a, 0x8000000080008081, 0x8000000000008080,
                                          0x0000000080000001, 0x8000000080008008 };

__device__ __forceinline__ uint64_t asm_cuda_keccak_ROTL64(const uint64_t x, const int offset) {
	uint64_t res;
	asm("{ // ROTL64 \n\t"
		".reg .u32 tl,th,vl,vh;\n\t"
		".reg .pred p;\n\t"
		"mov.b64 {tl,th}, %1;\n\t"
		"shf.l.wrap.b32 vl, tl, th, %2;\n\t"
		"shf.l.wrap.b32 vh, th, tl, %2;\n\t"
		"setp.lt.u32 p, %2, 32;\n\t"
		"@!p mov.b64 %0, {vl,vh};\n\t"
		"@p  mov.b64 %0, {vh,vl};\n\t"
	"}\n" : "=l"(res) : "l"(x) , "r"(offset)
	);
	return res;
}

__device__ __forceinline__ static void cuda_keccak_permutations(uint64_t* A)
{
    uint64_t *a00 = A, *a01 = A + 1, *a02 = A + 2, *a03 = A + 3, *a04 = A + 4;
    uint64_t *a05 = A + 5, *a06 = A + 6, *a07 = A + 7, *a08 = A + 8, *a09 = A + 9;
    uint64_t *a10 = A + 10, *a11 = A + 11, *a12 = A + 12, *a13 = A + 13, *a14 = A + 14;
    uint64_t *a15 = A + 15, *a16 = A + 16, *a17 = A + 17, *a18 = A + 18, *a19 = A + 19;
    uint64_t *a20 = A + 20, *a21 = A + 21, *a22 = A + 22, *a23 = A + 23, *a24 = A + 24;

    for (int i = 0; i < KECCAK_ROUND; i++) {

        /* Theta */
		uint64_t c0 = *a00^ *a05^ *a10^ *a15^ *a20;
		uint64_t c1 = *a01^ *a06^ *a11^ *a16^ *a21;
		uint64_t c2 = *a02^ *a07^ *a12^ *a17^ *a22;
		uint64_t c3 = *a03^ *a08^ *a13^ *a18^ *a23;
		uint64_t c4 =*a04^ *a09^ *a14^ *a19^ *a24;
		
        int64_t d1 = asm_cuda_keccak_ROTL64(c1, 1) ^ c4;
        int64_t d2 = asm_cuda_keccak_ROTL64(c2, 1) ^ c0;
        int64_t d3 = asm_cuda_keccak_ROTL64(c3, 1) ^ c1;
        int64_t d4 = asm_cuda_keccak_ROTL64(c4, 1) ^ c2;
        int64_t d0 = asm_cuda_keccak_ROTL64(c0, 1) ^ c3;

        *a00 ^= d1;
        *a05 ^= d1;
        *a10 ^= d1;
        *a15 ^= d1;
        *a20 ^= d1;
        *a01 ^= d2;
        *a06 ^= d2;
        *a11 ^= d2;
        *a16 ^= d2;
        *a21 ^= d2;
        *a02 ^= d3;
        *a07 ^= d3;
        *a12 ^= d3;
        *a17 ^= d3;
        *a22 ^= d3;
        *a03 ^= d4;
        *a08 ^= d4;
        *a13 ^= d4;
        *a18 ^= d4;
        *a23 ^= d4;
        *a04 ^= d0;
        *a09 ^= d0;
        *a14 ^= d0;
        *a19 ^= d0;
        *a24 ^= d0;


        /* Rho pi */
        c1 = asm_cuda_keccak_ROTL64(*a01, 1);
        *a01 = asm_cuda_keccak_ROTL64(*a06, 44);
        *a06 = asm_cuda_keccak_ROTL64(*a09, 20);
        *a09 = asm_cuda_keccak_ROTL64(*a22, 61);
        *a22 = asm_cuda_keccak_ROTL64(*a14, 39);
        *a14 = asm_cuda_keccak_ROTL64(*a20, 18);
        *a20 = asm_cuda_keccak_ROTL64(*a02, 62);
        *a02 = asm_cuda_keccak_ROTL64(*a12, 43);
        *a12 = asm_cuda_keccak_ROTL64(*a13, 25);
        *a13 = asm_cuda_keccak_ROTL64(*a19, 8);
        *a19 = asm_cuda_keccak_ROTL64(*a23, 56);
        *a23 = asm_cuda_keccak_ROTL64(*a15, 41);
        *a15 = asm_cuda_keccak_ROTL64(*a04, 27);
        *a04 = asm_cuda_keccak_ROTL64(*a24, 14);
        *a24 = asm_cuda_keccak_ROTL64(*a21, 2);
        *a21 = asm_cuda_keccak_ROTL64(*a08, 55);
        *a08 = asm_cuda_keccak_ROTL64(*a16, 45);
        *a16 = asm_cuda_keccak_ROTL64(*a05, 36);
        *a05 = asm_cuda_keccak_ROTL64(*a03, 28);
        *a03 = asm_cuda_keccak_ROTL64(*a18, 21);
        *a18 = asm_cuda_keccak_ROTL64(*a17, 15);
        *a17 = asm_cuda_keccak_ROTL64(*a11, 10);
        *a11 = asm_cuda_keccak_ROTL64(*a07, 6);
        *a07 = asm_cuda_keccak_ROTL64(*a10, 3);
        *a10 = c1;

        /* Chi * a ^ (~b) & c*/  
        c0 = *a00 ^ (~*a01 & *a02);  // use int2 vector this can be opt to 2 lop.b32 instruction
        c1 = *a01 ^ (~*a02 & *a03);
        *a02 ^= ~*a03 & *a04;
        *a03 ^= ~*a04 & *a00;
        *a04 ^= ~*a00 & *a01;
        *a00 = c0;
        *a01 = c1;

        c0 = *a05 ^ (~*a06 & *a07);
        c1 = *a06 ^ (~*a07 & *a08);
        *a07 ^= ~*a08 & *a09;
        *a08 ^= ~*a09 & *a05;
        *a09 ^= ~*a05 & *a06;
        *a05 = c0;
        *a06 = c1;

        c0 = *a10 ^ (~*a11 & *a12);
        c1 = *a11 ^ (~*a12 & *a13);
        *a12 ^= ~*a13 & *a14;
        *a13 ^= ~*a14 & *a10;
        *a14 ^= ~*a10 & *a11;
        *a10 = c0;
        *a11 = c1;

        c0 = *a15 ^ (~*a16 & *a17);
        c1 = *a16 ^ (~*a17 & *a18);
        *a17 ^= ~*a18 & *a19;
        *a18 ^= ~*a19 & *a15;
        *a19 ^= ~*a15 & *a16;
        *a15 = c0;
        *a16 = c1;

        c0 = *a20 ^ (~*a21 & *a22);
        c1 = *a21 ^ (~*a22 & *a23);
        *a22 ^= ~*a23 & *a24;
        *a23 ^= ~*a24 & *a20;
        *a24 ^= ~*a20 & *a21;
        *a20 = c0;
        *a21 = c1;

        /* Iota */
        *a00 ^= CUDA_KECCAK_CONSTS[i];
    }
}

__noinline__ __device__ static bool hashbelowtarget(const uint64_t *const __restrict__ hash, const uint64_t *const __restrict__ target)
{
    if (hash[3] > target[3])
        return false;
    if (hash[3] < target[3])
        return true;
    if (hash[2] > target[2])
        return false;
    if (hash[2] < target[2])
        return true;

    if (hash[1] > target[1])
        return false;
    if (hash[1] < target[1])
        return true;
    if (hash[0] > target[0])
        return false;

    return true;
}

__device__ uint64_t *addUint256(const uint64_t *a, const uint64_t b)
{
    uint64_t *result = new uint64_t[4];
    uint64_t sum = a[0] + b;
    result[0] = sum;

    uint64_t carry = (sum < a[0]) ? 1 : 0;
    for (int i = 1; i < 4; i++)
    {
        sum = a[i] + carry;
        result[i] = sum;
        carry = (sum < a[i]) ? 1 : 0;
    }

    return result;
}

__device__ void reverseArray(unsigned char *array, int n) {
    for (int i = 0; i < n / 2; ++i) {
        unsigned char temp = array[i];
        array[i] = array[n - 1 - i];
        array[n - 1 - i] = temp;
    }
}


extern "C" __global__ __launch_bounds__(1024, 1)
  void kernel_lilypad_pow(uint8_t* challenge, uint64_t* startNonce,  uint64_t* target, uint32_t n_batch, uint8_t* resNonce)
{
    uint32_t thread = blockIdx.x * blockDim.x + threadIdx.x; 
    if (thread >= n_batch) {
        return;
    }

       //increase nonce
    uint8_t* nonce = (uint8_t*)addUint256(startNonce, thread);
    uint64_t state[KECCAK_STATE_SIZE];
    memset(state, 0, sizeof(state));

    memcpy(state, challenge, 32);  // Copy challenge into state
    memcpy(state + 4, nonce, 32);  // Copy nonce into state starting from index 4

    state[8] ^= 1;
    state[16] ^= 9223372036854775808ULL; 

    cuda_keccak_permutations(state);

    uint8_t out[32];
    uint8_t* state_bytes = reinterpret_cast<uint8_t*>(state);
    #pragma unroll 32
    for (int i = 0;i<32; i++) {
        out[i] = state_bytes[31-i];
    }
    
    if (hashbelowtarget((uint64_t*)out, target)) {
        memcpy(resNonce, nonce, 32);
    } 

    delete nonce;//45
}


extern "C" __global__ __launch_bounds__(1024, 1)
  void kernel_lilypad_pow_debug(uint8_t* challenge, uint64_t* startNonce,  uint64_t* target, uint32_t n_batch, uint8_t* resNonce,  uint8_t *hash, uint8_t *pack)
{
    uint32_t thread = blockIdx.x * blockDim.x + threadIdx.x; 
    if (thread >= n_batch) {
        return;
    }

       //increase nonce
    uint8_t* nonce = (uint8_t*)addUint256(startNonce, thread);
    uint64_t state[KECCAK_STATE_SIZE];
    memset(state, 0, sizeof(state));

    memcpy(state, challenge, 32);  // Copy challenge into state
    memcpy(state + 4, nonce, 32);  // Copy nonce into state starting from index 4

    //uint8_t cuda_pack[64];
    //memcpy(cuda_pack, state, 64);

    state[8] ^= 1;
    state[16] ^= 9223372036854775808ULL; 

    cuda_keccak_permutations(state);

    uint8_t out[32];
    uint8_t* state_bytes = reinterpret_cast<uint8_t*>(state);
    #pragma unroll 32
    for (int i = 0;i<32; i++) {
        out[i] = state_bytes[31-i];
    }
    
    if (hashbelowtarget((uint64_t*)out, target)) {
       // reverseArray(out, 32);
       // memcpy(hash, out, 32);
       // memcpy(pack, cuda_pack, 64);
        memcpy(resNonce, nonce, 32);
    } 

    delete nonce;//45
}
