#include "hip/hip_runtime.h"
/*
 * keccak.cu  Implementation of Keccak/SHA3 digest
 *
 * Date: 12 June 2019
 * Revision: 1
 *
 * This file is released into the Public Domain.
 */
 
 
extern "C"
{
#include "keccak.cuh"
}

#define KECCAK_ROUND 24
#define KECCAK_STATE_SIZE 25
#define KECCAK_Q_SIZE 192

__constant__ LONG CUDA_KECCAK_CONSTS[24] = { 0x0000000000000001, 0x0000000000008082,
                                          0x800000000000808a, 0x8000000080008000, 0x000000000000808b, 0x0000000080000001, 0x8000000080008081,
                                          0x8000000000008009, 0x000000000000008a, 0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
                                          0x000000008000808b, 0x800000000000008b, 0x8000000000008089, 0x8000000000008003, 0x8000000000008002,
                                          0x8000000000000080, 0x000000000000800a, 0x800000008000000a, 0x8000000080008081, 0x8000000000008080,
                                          0x0000000080000001, 0x8000000080008008 };

typedef struct {

    BYTE sha3_flag;
    WORD digestbitlen;
    LONG rate_bits;
    LONG rate_BYTEs;
    LONG absorb_round;

    int64_t state[KECCAK_STATE_SIZE];
    BYTE q[KECCAK_Q_SIZE];

    LONG bits_in_queue;

} cuda_keccak_ctx_t;
typedef cuda_keccak_ctx_t CUDA_KECCAK_CTX;

__device__ LONG cuda_keccak_leuint64(void *in)
{
    LONG a;
    memcpy(&a, in, 8);
    return a;
}

__device__ int64_t cuda_keccak_MIN(int64_t a, int64_t b)
{
    if (a > b) return b;
    return a;
}

__device__ LONG cuda_keccak_UMIN(LONG a, LONG b)
{
    if (a > b) return b;
    return a;
}

__device__ void cuda_keccak_extract(cuda_keccak_ctx_t *ctx)
{
    LONG len = ctx->rate_bits >> 6;
    int64_t a;
    int s = sizeof(LONG);

    for (int i = 0;i < len;i++) {
        a = cuda_keccak_leuint64((int64_t*)&ctx->state[i]);
        memcpy(ctx->q + (i * s), &a, s);
    }
}

__device__ __forceinline__ LONG cuda_keccak_ROTL64(LONG a, LONG  b)
{
    return (a << b) | (a >> (64 - b));
}

__device__ void cuda_keccak_permutations(cuda_keccak_ctx_t * ctx)
{

    int64_t* A = ctx->state;;

    int64_t *a00 = A, *a01 = A + 1, *a02 = A + 2, *a03 = A + 3, *a04 = A + 4;
    int64_t *a05 = A + 5, *a06 = A + 6, *a07 = A + 7, *a08 = A + 8, *a09 = A + 9;
    int64_t *a10 = A + 10, *a11 = A + 11, *a12 = A + 12, *a13 = A + 13, *a14 = A + 14;
    int64_t *a15 = A + 15, *a16 = A + 16, *a17 = A + 17, *a18 = A + 18, *a19 = A + 19;
    int64_t *a20 = A + 20, *a21 = A + 21, *a22 = A + 22, *a23 = A + 23, *a24 = A + 24;

    for (int i = 0; i < KECCAK_ROUND; i++) {

        /* Theta */
        int64_t c0 = *a00 ^ *a05 ^ *a10 ^ *a15 ^ *a20;
        int64_t c1 = *a01 ^ *a06 ^ *a11 ^ *a16 ^ *a21;
        int64_t c2 = *a02 ^ *a07 ^ *a12 ^ *a17 ^ *a22;
        int64_t c3 = *a03 ^ *a08 ^ *a13 ^ *a18 ^ *a23;
        int64_t c4 = *a04 ^ *a09 ^ *a14 ^ *a19 ^ *a24;

        int64_t d1 = cuda_keccak_ROTL64(c1, 1) ^ c4;
        int64_t d2 = cuda_keccak_ROTL64(c2, 1) ^ c0;
        int64_t d3 = cuda_keccak_ROTL64(c3, 1) ^ c1;
        int64_t d4 = cuda_keccak_ROTL64(c4, 1) ^ c2;
        int64_t d0 = cuda_keccak_ROTL64(c0, 1) ^ c3;

        *a00 ^= d1;
        *a05 ^= d1;
        *a10 ^= d1;
        *a15 ^= d1;
        *a20 ^= d1;
        *a01 ^= d2;
        *a06 ^= d2;
        *a11 ^= d2;
        *a16 ^= d2;
        *a21 ^= d2;
        *a02 ^= d3;
        *a07 ^= d3;
        *a12 ^= d3;
        *a17 ^= d3;
        *a22 ^= d3;
        *a03 ^= d4;
        *a08 ^= d4;
        *a13 ^= d4;
        *a18 ^= d4;
        *a23 ^= d4;
        *a04 ^= d0;
        *a09 ^= d0;
        *a14 ^= d0;
        *a19 ^= d0;
        *a24 ^= d0;

        /* Rho pi */
        c1 = cuda_keccak_ROTL64(*a01, 1);
        *a01 = cuda_keccak_ROTL64(*a06, 44);
        *a06 = cuda_keccak_ROTL64(*a09, 20);
        *a09 = cuda_keccak_ROTL64(*a22, 61);
        *a22 = cuda_keccak_ROTL64(*a14, 39);
        *a14 = cuda_keccak_ROTL64(*a20, 18);
        *a20 = cuda_keccak_ROTL64(*a02, 62);
        *a02 = cuda_keccak_ROTL64(*a12, 43);
        *a12 = cuda_keccak_ROTL64(*a13, 25);
        *a13 = cuda_keccak_ROTL64(*a19, 8);
        *a19 = cuda_keccak_ROTL64(*a23, 56);
        *a23 = cuda_keccak_ROTL64(*a15, 41);
        *a15 = cuda_keccak_ROTL64(*a04, 27);
        *a04 = cuda_keccak_ROTL64(*a24, 14);
        *a24 = cuda_keccak_ROTL64(*a21, 2);
        *a21 = cuda_keccak_ROTL64(*a08, 55);
        *a08 = cuda_keccak_ROTL64(*a16, 45);
        *a16 = cuda_keccak_ROTL64(*a05, 36);
        *a05 = cuda_keccak_ROTL64(*a03, 28);
        *a03 = cuda_keccak_ROTL64(*a18, 21);
        *a18 = cuda_keccak_ROTL64(*a17, 15);
        *a17 = cuda_keccak_ROTL64(*a11, 10);
        *a11 = cuda_keccak_ROTL64(*a07, 6);
        *a07 = cuda_keccak_ROTL64(*a10, 3);
        *a10 = c1;

        /* Chi */
        c0 = *a00 ^ (~*a01 & *a02);
        c1 = *a01 ^ (~*a02 & *a03);
        *a02 ^= ~*a03 & *a04;
        *a03 ^= ~*a04 & *a00;
        *a04 ^= ~*a00 & *a01;
        *a00 = c0;
        *a01 = c1;

        c0 = *a05 ^ (~*a06 & *a07);
        c1 = *a06 ^ (~*a07 & *a08);
        *a07 ^= ~*a08 & *a09;
        *a08 ^= ~*a09 & *a05;
        *a09 ^= ~*a05 & *a06;
        *a05 = c0;
        *a06 = c1;

        c0 = *a10 ^ (~*a11 & *a12);
        c1 = *a11 ^ (~*a12 & *a13);
        *a12 ^= ~*a13 & *a14;
        *a13 ^= ~*a14 & *a10;
        *a14 ^= ~*a10 & *a11;
        *a10 = c0;
        *a11 = c1;

        c0 = *a15 ^ (~*a16 & *a17);
        c1 = *a16 ^ (~*a17 & *a18);
        *a17 ^= ~*a18 & *a19;
        *a18 ^= ~*a19 & *a15;
        *a19 ^= ~*a15 & *a16;
        *a15 = c0;
        *a16 = c1;

        c0 = *a20 ^ (~*a21 & *a22);
        c1 = *a21 ^ (~*a22 & *a23);
        *a22 ^= ~*a23 & *a24;
        *a23 ^= ~*a24 & *a20;
        *a24 ^= ~*a20 & *a21;
        *a20 = c0;
        *a21 = c1;

        /* Iota */
        *a00 ^= CUDA_KECCAK_CONSTS[i];
    }
}


__device__ void cuda_keccak_absorb(cuda_keccak_ctx_t *ctx, BYTE* in)
{

    LONG offset = 0;
    for (LONG i = 0; i < ctx->absorb_round; ++i) {
        ctx->state[i] ^= cuda_keccak_leuint64(in + offset);
        offset += 8;
    }

    cuda_keccak_permutations(ctx);
}

__device__ void cuda_keccak_pad(cuda_keccak_ctx_t *ctx)
{
    ctx->q[ctx->bits_in_queue >> 3] |= (1L << (ctx->bits_in_queue & 7));

    if (++(ctx->bits_in_queue) == ctx->rate_bits) {
        cuda_keccak_absorb(ctx, ctx->q);
        ctx->bits_in_queue = 0;
    }

    LONG full = ctx->bits_in_queue >> 6;
    LONG partial = ctx->bits_in_queue & 63;

    LONG offset = 0;
    for (int i = 0; i < full; ++i) {
        ctx->state[i] ^= cuda_keccak_leuint64(ctx->q + offset);
        offset += 8;
    }

    if (partial > 0) {
        LONG mask = (1L << partial) - 1;
        ctx->state[full] ^= cuda_keccak_leuint64(ctx->q + offset) & mask;
    }

    ctx->state[(ctx->rate_bits - 1) >> 6] ^= 9223372036854775808ULL;/* 1 << 63 */

    cuda_keccak_permutations(ctx);
    cuda_keccak_extract(ctx);

    ctx->bits_in_queue = ctx->rate_bits;
}

/*
 * Digestbitlen must be 128 224 256 288 384 512
 */
__device__ void cuda_keccak_init(cuda_keccak_ctx_t *ctx, WORD digestbitlen)
{
    memset(ctx, 0, sizeof(cuda_keccak_ctx_t));
    ctx->sha3_flag = 0;
    ctx->digestbitlen = digestbitlen;
    ctx->rate_bits = 1600 - ((ctx->digestbitlen) << 1);
    ctx->rate_BYTEs = ctx->rate_bits >> 3;
    ctx->absorb_round = ctx->rate_bits >> 6;
    ctx->bits_in_queue = 0;
}

/*
 * Digestbitlen must be 224 256 384 512
 */
__device__ void cuda_keccak_sha3_init(cuda_keccak_ctx_t *ctx, WORD digestbitlen)
{
    cuda_keccak_init(ctx, digestbitlen);
    ctx->sha3_flag = 1;
}

__device__ void cuda_keccak_update(cuda_keccak_ctx_t *ctx, BYTE *in, LONG inlen)
{
    int64_t BYTEs = ctx->bits_in_queue >> 3;
    int64_t count = 0;
    while (count < inlen) {
        if (BYTEs == 0 && count <= ((int64_t)(inlen - ctx->rate_BYTEs))) {
            do {
                cuda_keccak_absorb(ctx, in + count);
                count += ctx->rate_BYTEs;
            } while (count <= ((int64_t)(inlen - ctx->rate_BYTEs)));
        } else {
            int64_t partial = cuda_keccak_MIN(ctx->rate_BYTEs - BYTEs, inlen - count);
            memcpy(ctx->q + BYTEs, in + count, partial);

            BYTEs += partial;
            count += partial;

            if (BYTEs == ctx->rate_BYTEs) {
                cuda_keccak_absorb(ctx, ctx->q);
                BYTEs = 0;
            }
        }
    }
    ctx->bits_in_queue = BYTEs << 3;
}

__device__ void cuda_keccak_final(cuda_keccak_ctx_t *ctx, BYTE *out)
{
    if (ctx->sha3_flag) {
        int mask = (1 << 2) - 1;
        ctx->q[ctx->bits_in_queue >> 3] = (BYTE)(0x02 & mask);
        ctx->bits_in_queue += 2;
    }

    cuda_keccak_pad(ctx);
    LONG i = 0;

    while (i < ctx->digestbitlen) {
        if (ctx->bits_in_queue == 0) {
            cuda_keccak_permutations(ctx);
            cuda_keccak_extract(ctx);
            ctx->bits_in_queue = ctx->rate_bits;
        }

        LONG partial_block = cuda_keccak_UMIN(ctx->bits_in_queue, ctx->digestbitlen - i);
        memcpy(out + (i >> 3), ctx->q + (ctx->rate_BYTEs - (ctx->bits_in_queue >> 3)), partial_block >> 3);
        ctx->bits_in_queue -= partial_block;
        i += partial_block;
    }
}

extern "C" __global__ void kernel_keccak_hash(BYTE* indata, WORD inlen, BYTE* outdata, WORD n_batch, WORD KECCAK_BLOCK_SIZE)
{
    WORD thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread >= n_batch)
    {
        return;
    }
    BYTE* in = indata  + thread * inlen;
    BYTE* out = outdata  + thread * KECCAK_BLOCK_SIZE;
    CUDA_KECCAK_CTX ctx;
    cuda_keccak_init(&ctx, KECCAK_BLOCK_SIZE << 3);
    cuda_keccak_update(&ctx, in, inlen);
    cuda_keccak_final(&ctx, out);
}
