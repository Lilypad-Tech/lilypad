#include "hip/hip_runtime.h"
/*
 * keccak.cu  Implementation of Keccak/SHA3 digest
 *
 * Date: 12 June 2019
 * Revision: 1
 *
 * This file is released into the Public Domain.
 */
 
 
extern "C"
{
    #include "keccak.cuh"
}

#define KECCAK_ROUND 24
#define KECCAK_STATE_SIZE 25
#define KECCAK_Q_SIZE 192

__constant__ uint64_t CUDA_KECCAK_CONSTS[24] = { 0x0000000000000001, 0x0000000000008082,
                                          0x800000000000808a, 0x8000000080008000, 0x000000000000808b, 0x0000000080000001, 0x8000000080008081,
                                          0x8000000000008009, 0x000000000000008a, 0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
                                          0x000000008000808b, 0x800000000000008b, 0x8000000000008089, 0x8000000000008003, 0x8000000000008002,
                                          0x8000000000000080, 0x000000000000800a, 0x800000008000000a, 0x8000000080008081, 0x8000000000008080,
                                          0x0000000080000001, 0x8000000080008008 };


__constant__  uint64_t digestbitlen = 256;
__constant__  uint64_t rate_bits = 1088;
__constant__  uint64_t rate_BYTEs = 136;
__constant__  uint64_t absorb_round = 17;
typedef struct {
    int64_t state[KECCAK_STATE_SIZE];
    uint8_t q[KECCAK_Q_SIZE];

    uint64_t bits_in_queue;

} cuda_keccak_ctx_t;
typedef cuda_keccak_ctx_t CUDA_KECCAK_CTX;

__device__ uint64_t cuda_keccak_leuint64(void *in)
{
    uint64_t a;
    memcpy(&a, in, 8);
    return a;
}

__device__ int64_t cuda_keccak_MIN(int64_t a, int64_t b)
{
    if (a > b) return b;
    return a;
}

__device__ uint64_t cuda_keccak_UMIN(uint64_t a, uint64_t b)
{
    if (a > b) return b;
    return a;
}

__device__ void cuda_keccak_extract(cuda_keccak_ctx_t *ctx)
{
    uint64_t len = rate_bits >> 6;
    int64_t a;
    int s = sizeof(uint64_t);

    for (int i = 0;i < len;i++) {
        a = cuda_keccak_leuint64((int64_t*)&ctx->state[i]);
        memcpy(ctx->q + (i * s), &a, s);
    }
}
__device__ __forceinline__ uint64_t cuda_keccak_ROTL64(uint64_t a, uint64_t b) {
    return (a << b) | (a >> (64 - b));
}

__device__ void cuda_keccak_permutations(cuda_keccak_ctx_t *ctx) {
    int64_t* A = ctx->state;

    #pragma unroll 24
    for (int i = 0; i < KECCAK_ROUND; i++) {
        int64_t C[5], D[5];  

        // Theta
        C[0] = A[0] ^ A[5] ^ A[10] ^ A[15] ^ A[20];
        C[1] = A[1] ^ A[6] ^ A[11] ^ A[16] ^ A[21];
        C[2] = A[2] ^ A[7] ^ A[12] ^ A[17] ^ A[22];
        C[3] = A[3] ^ A[8] ^ A[13] ^ A[18] ^ A[23];
        C[4] = A[4] ^ A[9] ^ A[14] ^ A[19] ^ A[24];

        D[0] = cuda_keccak_ROTL64(C[1], 1) ^ C[4];
        D[1] = cuda_keccak_ROTL64(C[2], 1) ^ C[0];
        D[2] = cuda_keccak_ROTL64(C[3], 1) ^ C[1];
        D[3] = cuda_keccak_ROTL64(C[4], 1) ^ C[2];
        D[4] = cuda_keccak_ROTL64(C[0], 1) ^ C[3];

        #pragma unroll 25
        for (int j = 0; j < 25; j += 5) {
            A[j] ^= D[0];
            A[j + 1] ^= D[1];
            A[j + 2] ^= D[2];
            A[j + 3] ^= D[3];
            A[j + 4] ^= D[4];
        }

        // Rho Pi
        int64_t B[25];
        B[0] = A[0];
        B[1] = cuda_keccak_ROTL64(A[6], 44);
        B[2] = cuda_keccak_ROTL64(A[12], 43);
        B[3] = cuda_keccak_ROTL64(A[18], 21);
        B[4] = cuda_keccak_ROTL64(A[24], 14);
        B[5] = cuda_keccak_ROTL64(A[3], 28);
        B[6] = cuda_keccak_ROTL64(A[9], 20);
        B[7] = cuda_keccak_ROTL64(A[10], 3);
        B[8] = cuda_keccak_ROTL64(A[16], 45);
        B[9] = cuda_keccak_ROTL64(A[22], 61);
        B[10] = cuda_keccak_ROTL64(A[1], 1);
        B[11] = cuda_keccak_ROTL64(A[7], 6);
        B[12] = cuda_keccak_ROTL64(A[13], 25);
        B[13] = cuda_keccak_ROTL64(A[19], 8);
        B[14] = cuda_keccak_ROTL64(A[20], 18);
        B[15] = cuda_keccak_ROTL64(A[4], 27);
        B[16] = cuda_keccak_ROTL64(A[5], 36);
        B[17] = cuda_keccak_ROTL64(A[11], 10);
        B[18] = cuda_keccak_ROTL64(A[17], 15);
        B[19] = cuda_keccak_ROTL64(A[23], 56);
        B[20] = cuda_keccak_ROTL64(A[2], 62);
        B[21] = cuda_keccak_ROTL64(A[8], 55);
        B[22] = cuda_keccak_ROTL64(A[14], 39);
        B[23] = cuda_keccak_ROTL64(A[15], 41);
        B[24] = cuda_keccak_ROTL64(A[21], 2);

        // Chi
        #pragma unroll 24
        for (int j = 0; j < 25; j += 5) {
            #pragma unroll 5
            for (int k = 0; k < 5; ++k) {
                A[j + k] = B[j + k] ^ (~B[j + (k + 1) % 5] & B[j + (k + 2) % 5]);
            }
        }

        // Iota
        A[0] ^= CUDA_KECCAK_CONSTS[i];
    }
}


__device__ void cuda_keccak_absorb(cuda_keccak_ctx_t *ctx, uint8_t* in)
{

    uint64_t offset = 0;
    for (uint64_t i = 0; i < absorb_round; ++i) {//10
        ctx->state[i] ^= cuda_keccak_leuint64(in + offset);//18
        offset += 8;//9
    }

    cuda_keccak_permutations(ctx);//8
}

__device__ void cuda_keccak_pad(cuda_keccak_ctx_t *ctx)
{
    ctx->q[ctx->bits_in_queue >> 3] |= (1L << (ctx->bits_in_queue & 7)); //6

    if (++(ctx->bits_in_queue) == rate_bits) {//9
        cuda_keccak_absorb(ctx, ctx->q);//8
        ctx->bits_in_queue = 0;//53
    }

    uint64_t full = ctx->bits_in_queue >> 6;    //7
    uint64_t partial = ctx->bits_in_queue & 63; //8

    uint64_t offset = 0;
    for (int i = 0; i < full; ++i) {//52
        ctx->state[i] ^= cuda_keccak_leuint64(ctx->q + offset);//52
        offset += 8;//52
    }

    if (partial > 0) {//8
        uint64_t mask = (1L << partial) - 1;//17
        ctx->state[full] ^= cuda_keccak_leuint64(ctx->q + offset) & mask;//16
    }

    ctx->state[(rate_bits - 1) >> 6] ^= 9223372036854775808ULL;/* 1 << 63 */   //9

    cuda_keccak_permutations(ctx);//8
    cuda_keccak_extract(ctx);//58

    ctx->bits_in_queue = rate_bits;//37
}


/*
 * Digestbitlen must be 128 224 256 288 384 512
 */
__device__ void cuda_keccak_init(cuda_keccak_ctx_t *ctx)
{
    memset(ctx, 0, sizeof(cuda_keccak_ctx_t));
    ctx->bits_in_queue = 0;//11
}

__device__ void cuda_keccak_update(cuda_keccak_ctx_t *ctx, uint8_t *in, uint64_t inlen)
{
    int64_t BYTEs = ctx->bits_in_queue >> 3;
    int64_t count = 0;
    while (count < inlen) {//46
        if (BYTEs == 0 && count <= ((int64_t)(inlen - rate_BYTEs))) {//12
            do {
                cuda_keccak_absorb(ctx, in + count);//8
                count += rate_BYTEs;//56
            } while (count <= ((int64_t)(inlen - rate_BYTEs)));//46
        } else {
            int64_t partial = cuda_keccak_MIN(rate_BYTEs - BYTEs, inlen - count);//12
            memcpy(ctx->q + BYTEs, in + count, partial);//12

            BYTEs += partial;//10
            count += partial;//8

            if (BYTEs == rate_BYTEs) {//10
                cuda_keccak_absorb(ctx, ctx->q);//8
                BYTEs = 0;
            }
        }
    }
    ctx->bits_in_queue = BYTEs << 3;//8
}

__device__ void cuda_keccak_final_rev(cuda_keccak_ctx_t *ctx, uint8_t *out)
{
    cuda_keccak_pad(ctx);
    uint64_t i = 0;//6

    while (i < digestbitlen) {//46
        if (ctx->bits_in_queue == 0) {//9
            cuda_keccak_permutations(ctx);//8
            cuda_keccak_extract(ctx);//56
            ctx->bits_in_queue = rate_bits;//7
        }

        uint64_t partial_block = cuda_keccak_UMIN(ctx->bits_in_queue, digestbitlen - i);//9

        //directly reverse?
        int start = 31- (i >> 3);
        uint8_t* pos = ctx->q + (rate_BYTEs - (ctx->bits_in_queue >> 3));
        for (int j = 0; j< (partial_block >> 3); j++) {
            out[start-j] = pos[j];
        }

        ctx->bits_in_queue -= partial_block;//11
        i += partial_block;//11
    }
}



__noinline__ __device__ static bool hashbelowtarget(const uint64_t *const __restrict__ hash, const uint64_t *const __restrict__ target)
{
    if (hash[3] > target[3])//46
        return false;
    if (hash[3] < target[3])//46
        return true;
    if (hash[2] > target[2])//45
        return false;
    if (hash[2] < target[2])//45
        return true;

    if (hash[1] > target[1])//43
        return false;
    if (hash[1] < target[1])//43
        return true;
    if (hash[0] > target[0])//39
        return false;

    return true;
}

__device__ uint64_t *addUint256(const uint64_t *a, const uint64_t b)
{
    uint64_t *result = new uint64_t[4];//47
    uint64_t sum = a[0] + b;//10
    result[0] = sum;//10

    uint64_t carry = (sum < a[0]) ? 1 : 0;//12
    for (int i = 1; i < 4; i++)//13
    {
        sum = a[i] + carry;//16
        result[i] = sum;//14
        carry = (sum < a[i]) ? 1 : 0;//14
    }

    return result;
}
__device__ void reverse32BytesInPlace(uint8_t *data, uint8_t *out)
{
    for (int i = 0; i < 32; i++)//13
    {
       out[i] = data[31-i];
    }
}

extern "C" __global__ __launch_bounds__(1024)

  void kernel_lilypad_pow(uint8_t* chanllenge, uint64_t* startNonce,  uint64_t* target, uint64_t n_batch, uint8_t* resNonce)
{
    uint64_t thread = blockIdx.x * blockDim.x + threadIdx.x; //4
    if (thread >= n_batch) {//36
        return;
    }

    //pack input
    uint8_t in[64];
    memcpy(in, chanllenge, 32);
    //increase nonce
    uint8_t* nonce = (uint8_t*)addUint256(startNonce, thread);//35
    uint8_t nonce_rev[32];
    reverse32BytesInPlace(nonce, nonce_rev);//18
    memcpy(in+32, nonce_rev, 32);
    

    uint8_t out[32];
    CUDA_KECCAK_CTX ctx;
    cuda_keccak_init(&ctx);        //6
    cuda_keccak_update(&ctx, in,64);   //12
    cuda_keccak_final_rev(&ctx, out);       //6

    if (hashbelowtarget((uint64_t*)out, target)) {//49
        memcpy(resNonce, nonce_rev, 32);
    } 

    delete nonce;//45
}
